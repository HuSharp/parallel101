#include "hip/hip_runtime.h"
#include <cstdio>
#include <vector>
#include <memory>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "CudaAllocator.h"
#include "ticktock.h"

struct DisableCopy {
    DisableCopy() = default;
    DisableCopy(DisableCopy const &) = delete;
    DisableCopy &operator=(DisableCopy const &) = delete;
    DisableCopy(DisableCopy &&) = delete;
    DisableCopy &operator=(DisableCopy &&) = delete;
};

template <class T>
struct CudaArray : DisableCopy {
    hipArray *m_cuArray{};
    std::array<unsigned int, 3> m_dim{};

    struct BuildArgs {
        std::array<unsigned int, 3> const _dim;
        int flags = 0; // or hipArraySurfaceLoadStore
    };

    explicit CudaArray(BuildArgs _args) : m_dim(_args.dim) {
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<T>();  // or hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned)
        checkCudaErrors(hipMalloc3DArray(&m_cuArray, &channelDesc, make_hipExtent(m_dim[0], m_dim[1], m_dim[2]), _args.flags));
    }

    CudaArray &assign(T *_data) {
        hipMemcpy3DParms copy3DParams{};
        copy3DParams.srcPtr = make_hipPitchedPtr((void *)_data, m_dim[0] * sizeof(T), m_dim[1], m_dim[2]);
        copy3DParams.dstArray = m_cuArray;
        copy3DParams.extent = make_hipExtent(m_dim[0], m_dim[1], m_dim[2]);
        copy3DParams.kind = hipMemcpyHostToDevice;
        checkCudaErrors(hipMemcpy3D(&copy3DParams));
        return *this;
    }

    ~CudaArray() {
        checkCudaErrors(hipFreeArray(m_cuArray));
    }

    operator hipArray *() const {
        return m_cuArray;
    }
};

template <class T>
struct CudaSurface : DisableCopy {
    hipSurfaceObject_t m_cuSuf{};
    CudaArray<T> m_cuarr;

    explicit CudaSurface(typename CudaArray<T>::BuildArgs _cuarrArgs) : m_cuarr(_cuarrArgs) {
        hipResourceDesc resDesc{};
        resDesc.resType = hipResourceTypeArray;

        resDesc.res.array.array = m_cuarr.m_cuArray;
        hipCreateSurfaceObject(&m_cuSuf, &resDesc);
    }

    ~CudaTexture() {
        checkCudaErrors(hipDestroySurfaceObject(m_cuSuf));
    }

    CudaArray<T> &array() {
        return m_cuarr;
    }

    operator hipSurfaceObject_t() const {
        return m_cuSuf;
    }
};

template <class T>
struct CudaTexture : DisableCopy {
    hipTextureObject_t m_cuTex{};
    CudaArray<T> m_cuarr;

    explicit CudaTexture(typename CudaArray<T>::BuildArgs _cuarrArgs) : m_cuarr(_cuarrArgs) {
        hipResourceDesc resDesc{};
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = m_cuarr.m_cuArray;

        hipTextureDesc texDesc{};
        texDesc.addressMode[0] = hipAddressModeClamp; // or hipAddressModeWrap
        texDesc.addressMode[1] = hipAddressModeClamp; // or hipAddressModeWrap
        texDesc.addressMode[2] = hipAddressModeClamp; // or hipAddressModeWrap
        texDesc.filterMode = hipFilterModePoint;      // or hipFilterModeLinear
        texDesc.readMode = hipReadModeElementType;    // or hipReadModeNormalizedFloat
        texDesc.normalizedCoords = false;              // or true

        checkCudaErrors(hipCreateTextureObject(&m_cuTex, &resDesc, &texDesc, NULL));
    }

    ~CudaTexture() {
        checkCudaErrors(hipDestroyTextureObject(m_cuTex));
    }

    CudaArray<T> &array() {
        return m_cuarr;
    }

    operator hipTextureObject_t() const {
        return m_cuTex;
    }
};

__global__ void kernel(hipSurfaceObject_t out, hipTextureObject_t in) {
    int x = 0, y = 0;
    float fx = 0, fy = 0, fz = 0;
    float value = tex3D<float>(in, fx, fy, fz);
    value += 1;
    surf2Dwrite(value, out, x, y);
    // or hipBoundaryModeTrap, hipBoundaryModeClamp
}

int main() {
    CudaSurface<float> out({{1, 1, 1}, hipArraySurfaceLoadStore});
    CudaTexture<float> in({{1, 1, 1}, 0});
    return 0;
}
